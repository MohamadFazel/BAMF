#include "hip/hip_runtime.h"
/*!
 * \file GPUgaussMLEv2.cu
 * \author Keith Lidke
 * \date January 10, 2010
 * \brief This file contains all of the Cuda kernels.  The helper functions
 * are defined in GPUgaussLib.cuh
 */

#include "definitions.h"
#include "MatInvLib.h"
#include "GPUgaussLib.cuh"
#include "GPUgaussMLEv2.h"

//*******************************************************************************************
//theta is: {x,y,N,bg}
__global__ void kernel_MLEFit_XYNB_(const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param PSFSigma sigma of the point spread function
	 * \param sz nxn size of the subregion to fit
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
    //__shared__ float s_data[MEM];
    float M[NV_P*NV_P], Diag[NV_P], Minv[NV_P*NV_P];
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    const int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=NV_P;
    float dudt[NV_P];
    float d2udt2[NV_P];
    float NR_Numerator[NV_P], NR_Denominator[NV_P];
    float theta[NV_P];
    float maxjump[NV_P]={1e0f, 1e0f, 1e2f, 2e0f};
    float Nmax;

    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
	memset(M,0,NV_P*NV_P*sizeof(float));
	memset(Minv,0,NV_P*NV_P*sizeof(float));
    //load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
    //initial values
    kernel_CenterofMass2D(sz, s_data, &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*2*pi*PSFSigma*PSFSigma); //Added 2* on 8.9.16 to account for smoothing filter.
    
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
		memset(NR_Numerator,0,NV_P*sizeof(float));
		memset(NR_Denominator,0,NV_P*sizeof(float));

        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            PSFx=kernel_IntGauss1D(ii, theta[0], PSFSigma);
            PSFy=kernel_IntGauss1D(jj, theta[1], PSFSigma);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*jj+ii];
            
            //calculating derivatives
            kernel_DerivativeIntGauss1D(ii, theta[0], PSFSigma, theta[2], PSFy, &dudt[0], &d2udt2[0]);
            kernel_DerivativeIntGauss1D(jj, theta[1], PSFSigma, theta[2], PSFx, &dudt[1], &d2udt2[1]);
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0f;
            dudt[3] = 1.0f;
            d2udt2[3] = 0.0f;
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=data/model-1;
            if (model>10e-3f) df=data/pow(model, 2);
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
         
        // The update: Editted 8.9.16 to allow for larger jumps in Photons/BG
        theta[0]-=min(max(NR_Numerator[0]/NR_Denominator[0], -maxjump[0]), maxjump[0]);
        theta[1]-=min(max(NR_Numerator[1]/NR_Denominator[1], -maxjump[1]), maxjump[1]);
        theta[2]-=min(max(NR_Numerator[2]/NR_Denominator[2], -theta[2]), theta[2]);
        theta[3]-=min(max(NR_Numerator[3]/NR_Denominator[3], -theta[3]), theta[3]);

        // Any other constraints
        theta[2]=max(theta[2], 1.0f); //Make sure Photons is postitve
        theta[3]=max(theta[3], 0.01f); //Make sure Background is postitve
        
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        PSFx=kernel_IntGauss1D(ii, theta[0], PSFSigma);
        PSFy=kernel_IntGauss1D(jj, theta[1], PSFSigma);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*jj+ii];
        
        //calculating derivatives
        kernel_DerivativeIntGauss1D(ii, theta[0], PSFSigma, theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(jj, theta[1], PSFSigma, theta[2], PSFx, &dudt[1], NULL);
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if (model>0)
            if (data>0)Div+=data*log(model)-model-data*log(data)+data;
            else
                Div+=-model;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
    
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    
    return;
}

//*******************************************************************************************
__global__ void kernel_MLEFit_XYNBS_(const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param PSFSigma sigma of the point spread function
	 * \param sz nxn size of the subregion to fit
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
    
    //__shared__ float s_data[MEM];
    float M[NV_PS*NV_PS], Diag[NV_PS], Minv[NV_PS*NV_PS];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=NV_PS;
    float dudt[NV_PS];
    float d2udt2[NV_PS];
    float NR_Numerator[NV_PS], NR_Denominator[NV_PS];
    float theta[NV_PS];
    float maxjump[NV_PS]={1e0f, 1e0f, 1e2f, 2e0f, 5e-1f};
    float Nmax;
    
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
	memset(M,0,NV_PS*NV_PS*sizeof(float));
	memset(Minv,0,NV_PS*NV_PS*sizeof(float));      
    //load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
    
    //initial values
    kernel_CenterofMass2D(sz, s_data, &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*2*pi*PSFSigma*PSFSigma);
    theta[4]=PSFSigma;
    
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
  		memset(NR_Numerator,0,NV_PS*sizeof(float));
		memset(NR_Denominator,0,NV_PS*sizeof(float));
      
        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]);
            PSFy=kernel_IntGauss1D(jj, theta[1], theta[4]);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*jj+ii];
            
            //calculating derivatives
            kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], &d2udt2[0]);
            kernel_DerivativeIntGauss1D(jj, theta[1], theta[4], theta[2], PSFx, &dudt[1], &d2udt2[1]);
            kernel_DerivativeIntGauss2DSigma(ii, jj, theta[0], theta[1], theta[4], theta[2], PSFx, PSFy, &dudt[4], &d2udt2[4]);
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0f;
            dudt[3] = 1.0f;
            d2udt2[3] = 0.0f;
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=data/model-1;
            if (model>10e-3f) df=data/pow(model, 2);
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
        // The update: Editted 8.9.16 to allow for larger jumps in Photons/BG
        theta[0]-=min(max(NR_Numerator[0]/NR_Denominator[0], -maxjump[0]), maxjump[0]);
        theta[1]-=min(max(NR_Numerator[1]/NR_Denominator[1], -maxjump[1]), maxjump[1]);
        theta[2]-=min(max(NR_Numerator[2]/NR_Denominator[2], -theta[2]), theta[2]);
        theta[3]-=min(max(NR_Numerator[3]/NR_Denominator[3], -theta[3]), theta[3]);
        theta[4]-=min(max(NR_Numerator[4]/NR_Denominator[4], -theta[4]), theta[4]);

        // Any other constraints
        theta[2]=max(theta[2], 1.0f); //Make sure Photons is postitve
        theta[3]=max(theta[3], 0.01f); //Make sure Background is postitve
        theta[4]=max(theta[4], 0.5f); //Constrain Sigma
        theta[4]=min(theta[4], sz/2.0f); //Constrain Sigma
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0f;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]); //bug fix 8.9.16 
        PSFy=kernel_IntGauss1D(jj, theta[1], theta[4]); //bug fix 8.9.16 
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*jj+ii];
        
        //calculating derivatives
        kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(jj, theta[1], theta[4], theta[2], PSFx, &dudt[1], NULL);
        kernel_DerivativeIntGauss2DSigma(ii, jj, theta[0], theta[1], theta[4], theta[2], PSFx, PSFy, &dudt[4], NULL);
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if (model>0)
            if (data>0)Div+=data*log(model)-model-data*log(data)+data;
            else
                Div+=-model;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
  
    
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    
    return;
}

//*******************************************************************************************
__global__ void kernel_MLEFit_XYNBZ_(const float *d_data, const float PSFSigma_x, const float Ax, const float Ay, const float Bx, 
	const float By, const float gamma, const float d, const float PSFSigma_y, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param PSFSigma_x sigma of the point spread function on the x axis
	 * \param Ax ???
	 * \param Ay ???
	 * \param Bx ???
	 * \param By ???
	 * \param gamma ???
	 * \param d ???
	 * \param PSFSigma_y sigma of the point spread function on the y axis
	 * \param sz nxn size of the subregion to fit
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
    //__shared__ float s_data[MEM];
    float M[5*5], Diag[5], Minv[5*5];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=5;
    float dudt[5];
    float d2udt2[5];
    float NR_Numerator[5], NR_Denominator[5];
    float theta[5];
    float maxjump[5]={1e0f, 1e0f, 1e2f, 2e0f, 1e-1f};
    float Nmax;
    
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;

	memset(M,0,NV*NV*sizeof(float));
	memset(Minv,0,NV*NV*sizeof(float));      
    
    //load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);

    //initial values
    kernel_CenterofMass2D(sz, s_data, &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma_x, s_data, &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*pi*PSFSigma_x*PSFSigma_y*sqrt(2.0f));
    theta[4]=0;
   
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
  		memset(NR_Numerator,0,NV*sizeof(float));
		memset(NR_Denominator,0,NV*sizeof(float));
        
        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            kernel_DerivativeIntGauss2Dz(ii, jj, theta, PSFSigma_x,PSFSigma_y, Ax,Ay,Bx,By, gamma, d, &PSFx, &PSFy, dudt, d2udt2);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*jj+ii];
            
            //calculating remaining derivatives
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0f;
            dudt[3] = 1.0f;
            d2udt2[3] = 0.0f;
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=data/model-1;
            if (model>10e-3f) df=data/pow(model, 2);
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
        // The update: Editted 8.9.16 to allow for larger jumps in Photons/BG
        theta[0]-=min(max(NR_Numerator[0]/NR_Denominator[0], -maxjump[0]), maxjump[0]);
        theta[1]-=min(max(NR_Numerator[1]/NR_Denominator[1], -maxjump[1]), maxjump[1]);
        theta[2]-=min(max(NR_Numerator[2]/NR_Denominator[2], -theta[2]), theta[2]);
        theta[3]-=min(max(NR_Numerator[3]/NR_Denominator[3], -theta[3]), theta[3]);
        theta[4]-=min(max(NR_Numerator[4]/NR_Denominator[4], -maxjump[4]), maxjump[4]);
        
        // Any other constraints
        theta[2]=max(theta[2], 1.0f);
        theta[3]=max(theta[3], 0.01f);
        
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0f;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        
        kernel_DerivativeIntGauss2Dz(ii, jj, theta, PSFSigma_x,PSFSigma_y, Ax,Ay, Bx,By, gamma, d, &PSFx, &PSFy, dudt, NULL);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*jj+ii];
        
        //calculating remaining derivatives
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
       
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if (model>0)
            if (data>0)Div+=data*log(model)-model-data*log(data)+data;
            else
                Div+=-model;
    }
    
    // Matrix inverse (CRLB=F^-1) 
    kernel_MatInvN(M, Minv, Diag, NV);
  
   //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    return;
}

//*******************************************************************************************
__global__ void kernel_MLEFit_XYNBSXSY_(const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param PSFSigma sigma of the point spread function
	 * \param sz nxn size of the subregion to fit
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
 
    //__shared__ float s_data[MEM];
    float M[6*6], Diag[6], Minv[6*6];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=6;
    float dudt[6];
    float d2udt2[6];
    float NR_Numerator[6], NR_Denominator[6];
    float theta[6];
    float maxjump[6]={1e0f, 1e0f, 1e2f, 2e0f, 1e-1f,1e-1f};
    float Nmax;
    
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
	memset(M,0,NV*NV*sizeof(float));
	memset(Minv,0,NV*NV*sizeof(float));      
    
    //load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
    
	//initial values
    kernel_CenterofMass2D(sz, s_data, &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*2*pi*PSFSigma*PSFSigma);
    theta[4]=PSFSigma;
    theta[5]=PSFSigma;
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
  		memset(NR_Numerator,0,NV*sizeof(float));
		memset(NR_Denominator,0,NV*sizeof(float));
        
        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]);
            PSFy=kernel_IntGauss1D(jj, theta[1], theta[5]);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*jj+ii];
            
            //calculating derivatives
   
            kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], &d2udt2[0]);
            kernel_DerivativeIntGauss1D(jj, theta[1], theta[5], theta[2], PSFx, &dudt[1], &d2udt2[1]);
            kernel_DerivativeIntGauss1DSigma(ii, theta[0], theta[4], theta[2], PSFy, &dudt[4], &d2udt2[4]);
            kernel_DerivativeIntGauss1DSigma(jj, theta[1], theta[5], theta[2], PSFx, &dudt[5], &d2udt2[5]);
            
            
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0f;
            dudt[3] = 1.0f;
            d2udt2[3] = 0.0f;
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=data/model-1;
            if (model>10e-3f) df=data/pow(model, 2);
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
         // The update: Editted 8.9.16 to allow for larger jumps in Photons/BG
        theta[0]-=min(max(NR_Numerator[0]/NR_Denominator[0], -maxjump[0]), maxjump[0]);
        theta[1]-=min(max(NR_Numerator[1]/NR_Denominator[1], -maxjump[1]), maxjump[1]);
        theta[2]-=min(max(NR_Numerator[2]/NR_Denominator[2], -theta[2]), theta[2]);
        theta[3]-=min(max(NR_Numerator[3]/NR_Denominator[3], -theta[3]), theta[3]);
        theta[4]-=min(max(NR_Numerator[4]/NR_Denominator[4], -theta[4]), theta[4]);
        theta[5]-=min(max(NR_Numerator[5]/NR_Denominator[5], -theta[5]), theta[5]);

        // Any other constraints
        theta[2]=max(theta[2], 1.0f); //Make sure Photons is postitve
        theta[3]=max(theta[3], 0.01f); //Make sure Background is postitve
        theta[4]=max(theta[4], 0.5f); //Constrain Sigma
        theta[4]=min(theta[4], sz/2.0f); //Constrain SigmaX
        theta[5]=max(theta[5], 0.5f); //Constrain Sigma
        theta[5]=min(theta[5], sz/2.0f); //Constrain SigmaX
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0f;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        
        PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]);
        PSFy=kernel_IntGauss1D(jj, theta[1], theta[5]);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*jj+ii];
        
        //calculating derivatives
        kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(jj, theta[1], theta[5], theta[2], PSFx, &dudt[1], NULL);
        kernel_DerivativeIntGauss1DSigma(ii, theta[0], theta[4], theta[2], PSFy, &dudt[4], NULL);
        kernel_DerivativeIntGauss1DSigma(jj, theta[1], theta[5], theta[2], PSFx, &dudt[5], NULL);
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if (model>0)
            if (data>0)Div+=data*log(model)-model-data*log(data)+data;
            else
                Div+=-model;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
   
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    return;
}

// SCMOS Versions---------------------------------------

__global__ void kernel_MLEFit_SCMOSXYNB_(const float *d_data, const float *d_Coords, const float *d_GainRatio, 
	    const float PSFSigma, const int sz, const int Mapsz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param d_Coords array of subregions's pixel coordinates in original field of view.
	 * \param d_GainRatio calibration result of variance/gain^2 in each pixel of original field of view. 
	 * \param PSFSigma sigma of the point spread function
	 * \param sz nxn size of the subregion to fit
	 * \param Mapsz size of original field of view.
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
    //__shared__ float s_data[MEM];
    float M[NV_P*NV_P], Diag[NV_P], Minv[NV_P*NV_P];
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    const int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=NV_P;
    float dudt[NV_P];
    float d2udt2[NV_P];
    float NR_Numerator[NV_P], NR_Denominator[NV_P];
    float theta[NV_P];
    float maxjump[NV_P]={1e0f, 1e0f, 1e2f, 2e0f};
    float Nmax;
	float gainR;
	int GRind;
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
	memset(M,0,NV_P*NV_P*sizeof(float));
	memset(Minv,0,NV_P*NV_P*sizeof(float));
    //load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
	const float *s_Coords = d_Coords+(2*bx*BlockSize+2*tx);
	
    //initial values
    kernel_CenterofMass2D(sz, s_data, &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*2*pi*PSFSigma*PSFSigma);
    
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
		memset(NR_Numerator,0,NV_P*sizeof(float));
		memset(NR_Denominator,0,NV_P*sizeof(float));

        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            PSFx=kernel_IntGauss1D(ii, theta[0], PSFSigma);
            PSFy=kernel_IntGauss1D(jj, theta[1], PSFSigma);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*jj+ii];
            GRind=(s_Coords[1]+jj)*Mapsz+s_Coords[0]+ii;
			//GRind=(int)s_Coords[0];
			gainR=d_GainRatio[GRind];
            //calculating derivatives
            kernel_DerivativeIntGauss1D(ii, theta[0], PSFSigma, theta[2], PSFy, &dudt[0], &d2udt2[0]);//x
            kernel_DerivativeIntGauss1D(jj, theta[1], PSFSigma, theta[2], PSFx, &dudt[1], &d2udt2[1]);//y
            dudt[2] = PSFx*PSFy;// I
            d2udt2[2] = 0.0f;// I
            dudt[3] = 1.0f;// bg
            d2udt2[3] = 0.0f;// bg
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=(data-model)/(model+gainR); // add variance-gain ratio: v/g^2
            if (model>10e-3f) df=(data+gainR)/pow(model+gainR, 2); // add variance-gain ratio: v/g^2
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
        // The update: Editted 8.9.16 to allow for larger jumps in Photons/BG
        theta[0]-=min(max(NR_Numerator[0]/NR_Denominator[0], -maxjump[0]), maxjump[0]);
        theta[1]-=min(max(NR_Numerator[1]/NR_Denominator[1], -maxjump[1]), maxjump[1]);
        theta[2]-=min(max(NR_Numerator[2]/NR_Denominator[2], -theta[2]), theta[2]);
        theta[3]-=min(max(NR_Numerator[3]/NR_Denominator[3], -theta[3]), theta[3]);

        // Any other constraints
        theta[2]=max(theta[2], 1.0f); //Make sure Photons is postitve
        theta[3]=max(theta[3], 0.01f); //Make sure Background is postitve
        
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        PSFx=kernel_IntGauss1D(ii, theta[0], PSFSigma);
        PSFy=kernel_IntGauss1D(jj, theta[1], PSFSigma);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*jj+ii];
        
        //calculating derivatives
        kernel_DerivativeIntGauss1D(ii, theta[0], PSFSigma, theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(jj, theta[1], PSFSigma, theta[2], PSFx, &dudt[1], NULL);
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/(model+gainR);// add gain ratio
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if ((model+gainR)>0)
            if ((data+gainR)>0)Div+=(data+gainR)*log(model+gainR)-model-(data+gainR)*log(data+gainR)+data;// add gain ratio
            else
                Div+=-model-gainR;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
    
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    
    return;
}


//*******************************************************************************************
__global__ void kernel_MLEFit_SCMOSXYNBS_(const float *d_data, const float *d_Coords, const float *d_GainRatio,
	    const float PSFSigma, const int sz, const int Mapsz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param d_Coords array of subregions's pixel coordinates in original field of view.
	 * \param d_GainRatio calibration result of variance/gain^2 in each pixel of original field of view.
	 * \param PSFSigma sigma of the point spread function
	 * \param sz nxn size of the subregion to fit
	 * \param Mapsz size of original field of view.
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
    
    //__shared__ float s_data[MEM];
    float M[NV_PS*NV_PS], Diag[NV_PS], Minv[NV_PS*NV_PS];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=NV_PS;
    float dudt[NV_PS];
    float d2udt2[NV_PS];
    float NR_Numerator[NV_PS], NR_Denominator[NV_PS];
    float theta[NV_PS];
    float maxjump[NV_PS]={1e0f, 1e0f, 1e2f, 2e0f, 5e-1f};
    float Nmax;
    float gainR;
	int GRind;
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
	memset(M,0,NV_PS*NV_PS*sizeof(float));
	memset(Minv,0,NV_PS*NV_PS*sizeof(float));      
    //load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
    const float *s_Coords = d_Coords+(2*bx*BlockSize+2*tx);
    //initial values
    kernel_CenterofMass2D(sz, s_data, &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*2*pi*PSFSigma*PSFSigma);
    theta[4]=PSFSigma;
    
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
  		memset(NR_Numerator,0,NV_PS*sizeof(float));
		memset(NR_Denominator,0,NV_PS*sizeof(float));
      
        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]);
            PSFy=kernel_IntGauss1D(jj, theta[1], theta[4]);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*jj+ii];
            GRind=(s_Coords[1]+jj)*Mapsz+s_Coords[0]+ii;
			gainR=d_GainRatio[GRind];
            //calculating derivatives
            kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], &d2udt2[0]);
            kernel_DerivativeIntGauss1D(jj, theta[1], theta[4], theta[2], PSFx, &dudt[1], &d2udt2[1]);
            kernel_DerivativeIntGauss2DSigma(ii, jj, theta[0], theta[1], theta[4], theta[2], PSFx, PSFy, &dudt[4], &d2udt2[4]);
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0f;
            dudt[3] = 1.0f;
            d2udt2[3] = 0.0f;
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=(data-model)/(model+gainR);
            if (model>10e-3f) df=(data+gainR)/pow(model+gainR, 2);
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
        // The update: Editted 8.9.16 to allow for larger jumps in Photons/BG
        theta[0]-=min(max(NR_Numerator[0]/NR_Denominator[0], -maxjump[0]), maxjump[0]);
        theta[1]-=min(max(NR_Numerator[1]/NR_Denominator[1], -maxjump[1]), maxjump[1]);
        theta[2]-=min(max(NR_Numerator[2]/NR_Denominator[2], -theta[2]), theta[2]);
        theta[3]-=min(max(NR_Numerator[3]/NR_Denominator[3], -theta[3]), theta[3]);
        theta[4]-=min(max(NR_Numerator[4]/NR_Denominator[4], -theta[4]), theta[4]);

        // Any other constraints
        theta[2]=max(theta[2], 1.0f); //Make sure Photons is postitve
        theta[3]=max(theta[3], 0.01f); //Make sure Background is postitve
        theta[4]=max(theta[4], 0.5f); //Constrain Sigma
        theta[4]=min(theta[4], sz/2.0f); //Constrain Sigma
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0f;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]);
        PSFy=kernel_IntGauss1D(jj, theta[1], theta[4]);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*jj+ii];
        
        //calculating derivatives
        kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(jj, theta[1], theta[4], theta[2], PSFx, &dudt[1], NULL);
        kernel_DerivativeIntGauss2DSigma(ii, jj, theta[0], theta[1], theta[4], theta[2], PSFx, PSFy, &dudt[4], NULL);
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/(model+gainR);
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if ((model+gainR)>0)
            if ((data+gainR)>0)Div+=(data+gainR)*log(model+gainR)-model-(data+gainR)*log(data+gainR)+data;
            else
                Div+=-model-gainR;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
  
    
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    
    return;
}

//*******************************************************************************************
__global__ void kernel_MLEFit_SCMOSXYNBZ_(const float *d_data, const float *d_Coords, const float *d_GainRatio, const float *d_x0,
		const float PSFSigma_x, const float Ax, const float Ay, const float Bx, 
		const float By, const float gamma, const float d, const float PSFSigma_y, const int sz, const int Mapsz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param d_Coords array of subregions's pixel coordinates in original field of view.
	 * \param d_GainRatio calibration result of variance/gain^2 in each pixel of original field of view. 
	 * \param PSFSigma_x sigma of the point spread function on the x axis
	 * \param Ax ???
	 * \param Ay ???
	 * \param Bx ???
	 * \param By ???
	 * \param gamma ???
	 * \param d ???
	 * \param PSFSigma_y sigma of the point spread function on the y axis
	 * \param sz nxn size of the subregion to fit
	 * \param Mapsz size of original field of view.
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
    //__shared__ float s_data[MEM];
    float M[5*5], Diag[5], Minv[5*5];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=5;
    float dudt[5];
    float d2udt2[5];
    float NR_Numerator[5], NR_Denominator[5];
    float theta[5];
    float maxjump[5]={1e0f, 1e0f, 1e2f, 2e0f, 1e-1f};
    float Nmax;
    float gainR;
	int GRind;
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;

	memset(M,0,NV*NV*sizeof(float));
	memset(Minv,0,NV*NV*sizeof(float));      
    
    //load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
	const float *s_Coords = d_Coords+(2*bx*BlockSize+2*tx);
	const float *z_initial = d_x0+(bx*BlockSize+tx);
    //initial values
    kernel_CenterofMass2D(sz, s_data, &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma_x, s_data, &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*pi*PSFSigma_x*PSFSigma_y*sqrt(2.0f));
    theta[4]=z_initial[0];
   
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
  		memset(NR_Numerator,0,NV*sizeof(float));
		memset(NR_Denominator,0,NV*sizeof(float));
        
        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            kernel_DerivativeIntGauss2Dz(ii, jj, theta, PSFSigma_x,PSFSigma_y, Ax,Ay,Bx,By, gamma, d, &PSFx, &PSFy, dudt, d2udt2);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*jj+ii];
            GRind=(s_Coords[1]+jj)*Mapsz+s_Coords[0]+ii;
			gainR=d_GainRatio[GRind];
            //calculating remaining derivatives
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0f;
            dudt[3] = 1.0f;
            d2udt2[3] = 0.0f;
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=(data-model)/(model+gainR);
            if (model>10e-3f) df=(data+gainR)/pow(model+gainR, 2);
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
         // The update: Editted 8.9.16 to allow for larger jumps in Photons/BG
        theta[0]-=min(max(NR_Numerator[0]/NR_Denominator[0], -maxjump[0]), maxjump[0]);
        theta[1]-=min(max(NR_Numerator[1]/NR_Denominator[1], -maxjump[1]), maxjump[1]);
        theta[2]-=min(max(NR_Numerator[2]/NR_Denominator[2], -theta[2]), theta[2]);
        theta[3]-=min(max(NR_Numerator[3]/NR_Denominator[3], -theta[3]), theta[3]);
        theta[4]-=min(max(NR_Numerator[4]/NR_Denominator[4], -maxjump[4]), maxjump[4]);
        
        // Any other constraints
        theta[2]=max(theta[2], 1.0f);
        theta[3]=max(theta[3], 0.01f);
        
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0f;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        
        kernel_DerivativeIntGauss2Dz(ii, jj, theta, PSFSigma_x,PSFSigma_y, Ax,Ay, Bx,By, gamma, d, &PSFx, &PSFy, dudt, NULL);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*jj+ii];
        
        //calculating remaining derivatives
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
       
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if ((model+gainR)>0)
            if ((data+gainR)>0)Div+=(data+gainR)*log(model+gainR)-model-(data+gainR)*log(data+gainR)+data;
            else
                Div+=-model-gainR;
    }
    
    // Matrix inverse (CRLB=F^-1) 
    kernel_MatInvN(M, Minv, Diag, NV);
  
   //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    return;
}

//*******************************************************************************************
__global__ void kernel_MLEFit_SCMOSXYNBSXSY_(const float *d_data, const float *d_Coords, const float *d_GainRatio, 
	    const float PSFSigma, const int sz, const int Mapsz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	/*! 
	 * \brief basic MLE fitting kernel.  No additional parameters are computed.
	 * \param d_data array of subregions to fit copied to GPU
	 * \param d_Coords array of subregions's pixel coordinates in original field of view.
	 * \param d_GainRatio calibration result of variance/gain^2 in each pixel of original field of view. 
	 * \param PSFSigma sigma of the point spread function
	 * \param sz nxn size of the subregion to fit
	 * \param Mapsz size of original field of view.
	 * \param iterations number of iterations for solution to converge
	 * \param d_Parameters array of fitting parameters to return for each subregion
	 * \param d_CRLBs array of Cramer-Rao lower bound estimates to return for each subregion
	 * \param d_LogLikelihood array of loglikelihood estimates to return for each subregion
	 * \param Nfits number of subregions to fit
	 */
 
    //__shared__ float s_data[MEM];
    float M[6*6], Diag[6], Minv[6*6];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll;
    float model, cf, df, data;
    float Div;
    float PSFy, PSFx;
    int NV=6;
    float dudt[6];
    float d2udt2[6];
    float NR_Numerator[6], NR_Denominator[6];
    float theta[6];
    float maxjump[6]={1e0f, 1e0f, 1e2f, 2e0f, 1e-1f,1e-1f};
    float Nmax;
    float gainR;
	int GRind;
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
	memset(M,0,NV*NV*sizeof(float));
	memset(Minv,0,NV*NV*sizeof(float));      
    
    //load data
    const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
    const float *s_Coords = d_Coords+(2*bx*BlockSize+2*tx);
	//initial values
    kernel_CenterofMass2D(sz, s_data, &theta[0], &theta[1]);
    kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &theta[3]);
    theta[2]=max(0.0f, (Nmax-theta[3])*2*2*pi*PSFSigma*PSFSigma);
    theta[4]=PSFSigma;
    theta[5]=PSFSigma;
    for (kk=0;kk<iterations;kk++) {//main iterative loop
        
        //initialize
  		memset(NR_Numerator,0,NV*sizeof(float));
		memset(NR_Denominator,0,NV*sizeof(float));
        
        for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
            PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]);
            PSFy=kernel_IntGauss1D(jj, theta[1], theta[5]);
            
            model=theta[3]+theta[2]*PSFx*PSFy;
            data=s_data[sz*jj+ii];
            GRind=(s_Coords[1]+jj)*Mapsz+s_Coords[0]+ii;
			gainR=d_GainRatio[GRind];
            //calculating derivatives
            kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], &d2udt2[0]);
            kernel_DerivativeIntGauss1D(jj, theta[1], theta[5], theta[2], PSFx, &dudt[1], &d2udt2[1]);
            kernel_DerivativeIntGauss1DSigma(ii, theta[0], theta[4], theta[2], PSFy, &dudt[4], &d2udt2[4]);
            kernel_DerivativeIntGauss1DSigma(jj, theta[1], theta[5], theta[2], PSFx, &dudt[5], &d2udt2[5]);
            dudt[2] = PSFx*PSFy;
            d2udt2[2] = 0.0f;
            dudt[3] = 1.0f;
            d2udt2[3] = 0.0f;
            
            cf=0.0f;
            df=0.0f;
            if (model>10e-3f) cf=(data-model)/(model+gainR);
            if (model>10e-3f) df=(data+gainR)/pow(model+gainR, 2);
            cf=min(cf, 10e4f);
            df=min(df, 10e4f);
            
            for (ll=0;ll<NV;ll++){
                NR_Numerator[ll]+=dudt[ll]*cf;
                NR_Denominator[ll]+=d2udt2[ll]*cf-pow(dudt[ll], 2)*df;
            }
        }
        
         // The update: Editted 8.9.16 to allow for larger jumps in Photons/BG
        theta[0]-=min(max(NR_Numerator[0]/NR_Denominator[0], -maxjump[0]), maxjump[0]);
        theta[1]-=min(max(NR_Numerator[1]/NR_Denominator[1], -maxjump[1]), maxjump[1]);
        theta[2]-=min(max(NR_Numerator[2]/NR_Denominator[2], -theta[2]), theta[2]);
        theta[3]-=min(max(NR_Numerator[3]/NR_Denominator[3], -theta[3]), theta[3]);
        theta[4]-=min(max(NR_Numerator[4]/NR_Denominator[4], -theta[4]), theta[4]);
        theta[5]-=min(max(NR_Numerator[5]/NR_Denominator[5], -theta[5]), theta[5]);

        // Any other constraints
        theta[2]=max(theta[2], 1.0f); //Make sure Photons is postitve
        theta[3]=max(theta[3], 0.01f); //Make sure Background is postitve
        theta[4]=max(theta[4], 0.5f); //Constrain Sigma
        theta[4]=min(theta[4], sz/2.0f); //Constrain SigmaX
        theta[5]=max(theta[5], 0.5f); //Constrain Sigma
        theta[5]=min(theta[5], sz/2.0f); //Constrain SigmaX
    }
    
    // Calculating the CRLB and LogLikelihood
    Div=0.0f;
    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
        
        PSFx=kernel_IntGauss1D(ii, theta[0], theta[4]);
        PSFy=kernel_IntGauss1D(jj, theta[1], theta[5]);
        
        model=theta[3]+theta[2]*PSFx*PSFy;
        data=s_data[sz*jj+ii];
        
        //calculating derivatives
        kernel_DerivativeIntGauss1D(ii, theta[0], theta[4], theta[2], PSFy, &dudt[0], NULL);
        kernel_DerivativeIntGauss1D(jj, theta[1], theta[5], theta[2], PSFx, &dudt[1], NULL);
        kernel_DerivativeIntGauss1DSigma(ii, theta[0], theta[4], theta[2], PSFy, &dudt[4], NULL);
        kernel_DerivativeIntGauss1DSigma(jj, theta[1], theta[5], theta[2], PSFx, &dudt[5], NULL);
        dudt[2] = PSFx*PSFy;
        dudt[3] = 1.0f;
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= dudt[ll]*dudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if ((model+gainR)>0)
            if ((data+gainR)>0)Div+=(data+gainR)*log(model+gainR)-model-(data+gainR)*log(data+gainR)+data;
            else
                Div+=-model-gainR;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
   
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=theta[kk];
    for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
    d_LogLikelihood[BlockSize*bx+tx] = Div;
    return;
}

