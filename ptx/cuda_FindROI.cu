#include "hip/hip_runtime.h"

//#include "hip/hip_runtime.h"
#include "definitions.h"
//#include "kernel.h"

//#include <stdio.h>
//#include <stdlib.h>
//#include <string.h>

__global__ void kernel_gaussX(float * d, int Xsize, float b0, float b1, float b2, float b3, float B)
{
    
	//this kernel does gaussian filter along the X dimension.  
	int Ysize = blockDim.x;
	int idy = threadIdx.x;
	int idz = blockIdx.x;

	float w0, w1, w2, w3;
	float temp;
	int ii = 0;
	const int base = idz*Xsize*Ysize + idy*Xsize;

	//forward
	w1 = w2 = w3 = d[base];
	for (ii = 0; ii<Xsize; ii++)
	{
		w0 = d[base + ii];
		temp = w0*B + (b1*w1 + b2*w2 + b3*w3) / b0;
		d[base + ii] = temp;
		w3 = w2;
		w2 = w1;
		w1 = temp;
	}

	//backward
	w1 = w2 = w3 = d[base + Xsize - 1];
	for (ii = Xsize - 1; ii >= 0; ii--)
	{
		w0 = d[base + ii];
		temp = w0*B + (b1*w1 + b2*w2 + b3*w3) / b0;
		d[base + ii] = temp;
		w3 = w2;
		w2 = w1;
		w1 = temp;
	}
}

__global__ void kernel_gaussY(float * d, int Ysize, float b0, float b1, float b2, float b3, float B)
{
	//this kernel does gaussian filter along the Y dimension.  
	int Xsize = blockDim.x;
	int idx = threadIdx.x;
	int idz = blockIdx.x;

	float w0, w1, w2, w3;
	float temp;
	int ii = 0;
	const int base = idz*Xsize*Ysize + idx;

	//forward
	w1 = w2 = w3 = d[base];
	for (ii = 0; ii<Ysize; ii++)
	{
		w0 = d[base + ii*Xsize];
		temp = w0*B + (b1*w1 + b2*w2 + b3*w3) / b0;
		d[base + ii*Xsize] = temp;
		w3 = w2;
		w2 = w1;
		w1 = temp;
	}

	//backward
	w1 = w2 = w3 = d[base + Xsize*(Ysize - 1)];
	for (ii = Ysize - 1; ii >= 0; ii--)
	{
		w0 = d[base + ii*Xsize];
		temp = w0*B + (b1*w1 + b2*w2 + b3*w3) / b0;
		d[base + ii*Xsize] = temp;
		w3 = w2;
		w2 = w1;
		w1 = temp;
	}
}

__global__ void kernel_subtract(float * d_A, float * d_B)
{
	//this kernel does gaussian filter along the Y dimension.  
	int Xsize = blockDim.x;
	int Ysize = gridDim.x;
	int idx = threadIdx.x + Xsize*blockIdx.x + Xsize*Ysize*blockIdx.y;
	d_A[idx] = d_A[idx] - d_B[idx];
}

__global__ void kernel_maxX(float * d_A, float * d_B, int kernelsz, float minval)
{
	//this kernel does max finding along the X dimension.  
	int Xsize = blockDim.x;
	int Ysize = gridDim.x;
	
	//this is the pixel that we are searching around
	int idx = threadIdx.x + Xsize*blockIdx.x + Xsize*Ysize*blockIdx.y;
	int x = threadIdx.x;

	//define search only up to edges 
	int start = fmaxf(0, x - kernelsz) + Xsize*blockIdx.x + Xsize*Ysize*blockIdx.y;
	int end = fminf(Xsize - 1, x + kernelsz) + Xsize*blockIdx.x + Xsize*Ysize*blockIdx.y;

	float maxval = minval;
	float inpixel = d_A[idx];

	for (int ii = start; ii<end + 1; ii++) 
		maxval = fmaxf(maxval, d_A[ii]);

	//if any other pixel is larger set pixel idx to negative of that value, otherwise keep
	d_B[idx] = (maxval>inpixel)*-maxval + (maxval == inpixel)*maxval;

}

__global__ void kernel_maxY(float * d_A, float * d_B, int kernelsz, float minval)
{
	//this kernel does max finding in second dimension.
	int Ysize = blockDim.x;
	int Xsize = gridDim.x;

	int x = blockIdx.x;
	int y = threadIdx.x;
	int z = blockIdx.y;

	//this is the pixel that we are searching around
	int idx = x + Xsize*y + Xsize*Ysize*z;

	//define search only up to edges 
	int start = fmaxf(0, y - kernelsz);
	int end = fminf(Ysize - 1, y + kernelsz);

	float maxval = minval;
	float inpixel = d_B[idx];

	//find the maximum absolute value in the filter window
	for (int ii = start; ii<=end; ii++)
		maxval = fmaxf(maxval, fabsf(d_B[ii*Xsize + x + Xsize*Ysize*z]));

	//if our pixel under test is equal to maximum, then flag that with '1', otherwise '0'
	d_A[idx] = fabsf(maxval - inpixel)< 1e-6;
	

}

